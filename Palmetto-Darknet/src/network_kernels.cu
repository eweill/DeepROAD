#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "deconvolutional_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"
}

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);

void forward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }
        if(l.type == CONVOLUTIONAL){
            forward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            forward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            forward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            forward_local_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            forward_detection_layer_gpu(l, state);
        } else if(l.type == REGION){
            forward_region_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            forward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            forward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            forward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            forward_crnn_layer_gpu(l, state);
        } else if(l.type == CROP){
            forward_crop_layer_gpu(l, state);
        } else if(l.type == COST){
            forward_cost_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            forward_softmax_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            forward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            forward_batchnorm_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            forward_maxpool_layer_gpu(l, state);
        } else if(l.type == REORG){
            forward_reorg_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            forward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            forward_dropout_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            forward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            forward_shortcut_layer_gpu(l, state);
        }
        state.input = l.output_gpu;
    }
}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        if(l.type == CONVOLUTIONAL){
            backward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            backward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            backward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            backward_local_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            if(i != 0) backward_maxpool_layer_gpu(l, state);
        } else if(l.type == REORG){
            backward_reorg_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            if(i != 0) backward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            backward_dropout_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            backward_detection_layer_gpu(l, state);
        } else if(l.type == REGION){
            backward_region_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            backward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            backward_batchnorm_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            if(i != 0) backward_softmax_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            backward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            backward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            backward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            backward_crnn_layer_gpu(l, state);
        } else if(l.type == COST){
            backward_cost_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            backward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            backward_shortcut_layer_gpu(l, state);
        }
    }
}

void update_network_gpu(network net)
{
    int i;
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        if(l.type == CONVOLUTIONAL){
            update_convolutional_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == DECONVOLUTIONAL){
            update_deconvolutional_layer_gpu(l, rate, net.momentum, net.decay);
        } else if(l.type == CONNECTED){
            update_connected_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == GRU){
            update_gru_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == RNN){
            update_rnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == CRNN){
            update_crnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == LOCAL){
            update_local_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}

void forward_backward_network_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
    forward_network_gpu(net, state);
    backward_network_gpu(net, state);
}

float train_network_datum_gpu(network net, float *x, float *y)
{
    forward_backward_network_gpu(net, x, y);
    float error = get_network_cost(net);
    if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);

    return error;
}

typedef struct {
    network net;
    float *X;
    float *y;
} train_args;

void *train_thread(void *ptr)
{
    train_args args = *(train_args*)ptr;

    hipError_t status = hipSetDevice(args.net.gpu_index);
    check_error(status);
    forward_backward_network_gpu(args.net, args.X, args.y);
    free(ptr);
    return 0;
}

pthread_t train_network_in_thread(train_args args)
{
    pthread_t thread;
    train_args *ptr = (train_args *)calloc(1, sizeof(train_args));
    *ptr = args;
    if(pthread_create(&thread, 0, train_thread, ptr)) error("Thread creation failed");
    return thread;
}

float train_networks(network *nets, int n, data d)
{
    int batch = nets[0].batch;
    float **X = (float **) calloc(n, sizeof(float *));
    float **y = (float **) calloc(n, sizeof(float *));
    pthread_t *threads = (pthread_t *) calloc(n, sizeof(pthread_t));

    int i;
    float sum = 0;
    for(i = 0; i < n; ++i){
        X[i] = (float *) calloc(batch*d.X.cols, sizeof(float));
        y[i] = (float *) calloc(batch*d.y.cols, sizeof(float));
        get_next_batch(d, batch, i*batch, X[i], y[i]);
        float err = train_network_datum(nets[i], X[i], y[i]);
        sum += err;
    }
    free(X);
    free(y);
    return (float)sum/(n*batch);
}

float *get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    state.input = cuda_make_array(input, size);
    state.truth = 0;
    state.train = 0;
    state.delta = 0;
    forward_network_gpu(net, state);
    float *out = get_network_output_gpu(net);
    cuda_free(state.input);
    return out;
}

