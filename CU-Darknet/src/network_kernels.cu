#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "deconvolutional_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"
}

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);

void forward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }
        if(l.type == CONVOLUTIONAL){
            forward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            forward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            forward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            forward_local_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            forward_detection_layer_gpu(l, state);
        } else if(l.type == REGION){
            forward_region_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            forward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            forward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            forward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            forward_crnn_layer_gpu(l, state);
        } else if(l.type == CROP){
            forward_crop_layer_gpu(l, state);
        } else if(l.type == COST){
            forward_cost_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            forward_softmax_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            forward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            forward_batchnorm_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            forward_maxpool_layer_gpu(l, state);
        } else if(l.type == REORG){
            forward_reorg_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            forward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            forward_dropout_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            forward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            forward_shortcut_layer_gpu(l, state);
        }
        state.input = l.output_gpu;
    }
}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        if(l.type == CONVOLUTIONAL){
            backward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            backward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            backward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            backward_local_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            if(i != 0) backward_maxpool_layer_gpu(l, state);
        } else if(l.type == REORG){
            backward_reorg_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            if(i != 0) backward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            backward_dropout_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            backward_detection_layer_gpu(l, state);
        } else if(l.type == REGION){
            backward_region_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            backward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            backward_batchnorm_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            if(i != 0) backward_softmax_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            backward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            backward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            backward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            backward_crnn_layer_gpu(l, state);
        } else if(l.type == COST){
            backward_cost_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            backward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            backward_shortcut_layer_gpu(l, state);
        }
    }
}

void update_network_gpu(network net)
{
    int i;
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        if(l.type == CONVOLUTIONAL){
            update_convolutional_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == DECONVOLUTIONAL){
            update_deconvolutional_layer_gpu(l, rate, net.momentum, net.decay);
        } else if(l.type == CONNECTED){
            update_connected_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == GRU){
            update_gru_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == RNN){
            update_rnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == CRNN){
            update_crnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == LOCAL){
            update_local_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}

void forward_backward_network_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
    forward_network_gpu(net, state);
    backward_network_gpu(net, state);
}

float train_network_datum_gpu(network net, float *x, float *y)
{
    *net.seen += net.batch;
    forward_backward_network_gpu(net, x, y);
    float error = get_network_cost(net);
    if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);

    return error;
}

typedef struct {
    network net;
    float *X;
    float *y;
} train_args;

void *train_thread(void *ptr)
{
    train_args args = *(train_args*)ptr;

    cuda_set_device(args.net.gpu_index);
    forward_backward_network_gpu(args.net, args.X, args.y);
    free(ptr);
    return 0;
}

pthread_t train_network_in_thread(network net, float *X, float *y)
{
    pthread_t thread;
    train_args *ptr = (train_args *)calloc(1, sizeof(train_args));
    ptr->net = net;
    ptr->X = X;
    ptr->y = y;
    if(pthread_create(&thread, 0, train_thread, ptr)) error("Thread creation failed");
    return thread;
}

void pull_updates(layer l)
{
#ifdef GPU
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.n*l.size*l.size*l.c);
        if(l.scale_updates) cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
#endif
}

void push_updates(layer l)
{
#ifdef GPU
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.n*l.size*l.size*l.c);
        if(l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
#endif
}

void merge_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.n*l.size*l.size*l.c, 1, l.weight_updates, 1, base.weight_updates, 1);
        if (l.scale_updates) {
            axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
    }
}

void distribute_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        copy_cpu(l.n, base.bias_updates, 1, l.bias_updates, 1);
        copy_cpu(l.n*l.size*l.size*l.c, base.weight_updates, 1, l.weight_updates, 1);
        if (l.scale_updates) {
            copy_cpu(l.n, base.scale_updates, 1, l.scale_updates, 1);
        }
    } else if(l.type == CONNECTED) {
        copy_cpu(l.outputs, base.bias_updates, 1, l.bias_updates, 1);
        copy_cpu(l.outputs*l.inputs, base.weight_updates, 1, l.weight_updates, 1);
    }
}

void sync_updates(network *nets, int n)
{
    int i,j;
    int layers = nets[0].n;
    network net = nets[0];
    for (j = 0; j < layers; ++j) {
        layer base = net.layers[j];
        cuda_set_device(net.gpu_index);
        pull_updates(base);
        for (i = 1; i < n; ++i) {
            cuda_set_device(nets[i].gpu_index);
            layer l = nets[i].layers[j];
            pull_updates(l);
            merge_updates(l, base);
        }
        for (i = 1; i < n; ++i) {
            cuda_set_device(nets[i].gpu_index);
            layer l = nets[i].layers[j];
            distribute_updates(l, base);
            push_updates(l);
        }
        cuda_set_device(net.gpu_index);
        push_updates(base);
    }
    for (i = 0; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        if(i > 0) nets[i].momentum = 0;
        update_network_gpu(nets[i]);
    }
}

float train_networks(network *nets, int n, data d)
{
    int batch = nets[0].batch;
    assert(batch * n == d.X.rows);
    assert(nets[0].subdivisions % n == 0);
    float **X = (float **) calloc(n, sizeof(float *));
    float **y = (float **) calloc(n, sizeof(float *));
    pthread_t *threads = (pthread_t *) calloc(n, sizeof(pthread_t));

    int i;
    float sum = 0;
    for(i = 0; i < n; ++i){
        X[i] = (float *) calloc(batch*d.X.cols, sizeof(float));
        y[i] = (float *) calloc(batch*d.y.cols, sizeof(float));
        get_next_batch(d, batch, i*batch, X[i], y[i]);
        threads[i] = train_network_in_thread(nets[i], X[i], y[i]);
    }
    for(i = 0; i < n; ++i){
        pthread_join(threads[i], 0);
        *nets[i].seen += n*nets[i].batch;
        printf("%f\n", get_network_cost(nets[i]) / batch);
        sum += get_network_cost(nets[i]);
        free(X[i]);
        free(y[i]);
    }
    if (((*nets[0].seen) / nets[0].batch) % nets[0].subdivisions == 0) sync_updates(nets, n);
    free(X);
    free(y);
    free(threads);
    return (float)sum/(n*batch);
}

float *get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    state.input = cuda_make_array(input, size);
    state.truth = 0;
    state.train = 0;
    state.delta = 0;
    forward_network_gpu(net, state);
    float *out = get_network_output_gpu(net);
    cuda_free(state.input);
    return out;
}

